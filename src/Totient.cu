
#include <hip/hip_runtime.h>
// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

//Kernel function to compute the Totient value. It just takes the number on which the
//totient should be computed and computes the totient for the current thread number +
//size of the totals threads the GPU can create.
extern "C" __global__ void ComputeTotient(unsigned long long int N){

    //Variable declarations
    int thr, size, rank;
    unsigned long long int count=0, b, temp, a;

    //Rank and size computations
    thr = threadIdx.x;
    size = gridDim.x*NT;
    rank = blockIdx.x*NT + thr;

    //loop to compute the totient for current thread rank
    for (unsigned long long int x = rank; x < N; x += size){
    a=x;
    b=N;
        while (b > 0){
            temp = b;
            b = a % b;
            a = temp;
        }

        if(a==1)
        ++count;
    }

 // assigning the counter value to the shared memory variable
    shrCount[thr] = count;
    __syncthreads();

    //reduction of the shared variables in parallel
    for (int i = NT/2; i > 0; i >>= 1){
        if (thr < i)
        shrCount[thr] += shrCount[thr+i];
        __syncthreads();
    }

    // Adding the total of each block to global variable
    if (thr == 0)
    atomicAdd (&devCount, shrCount[0]);
}
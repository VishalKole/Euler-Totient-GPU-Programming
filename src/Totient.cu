#include "hip/hip_runtime.h"
// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

extern "C" __global__ void ComputeTotient(unsigned long long int N){

    int thr, size, rank;
    unsigned long long int count=0, b, temp;

    size = gridDim.x*NT;
    rank = blockIdx.x*NT + threadIdx.x;

    for (unsigned long long int a = rank; a < N; a += size){
    b=N;

    //    while(a != b){
    //            if( a > b){
    //               a = a - b;}
    //            else{
    //               b = b - a;}
    //            }

        while (b != 0){
            temp = b;
            b = a % b;
            a = temp;
        }


        if(a==1){
        ++count;}
    }

 // Shared memory parallel reduction within thread block.
    shrCount[thr] = count;
    syncthreads();

    for (int i = NT/2; i > 0; i >>= 1){

        if (thr < i)
        shrCount[thr] += shrCount[thr+i];
        syncthreads();
    }

    // Atomic reduction into overall counter.
    if (thr == 0)
    atomicAdd (&devCount, shrCount[0]);

}
#include "hip/hip_runtime.h"
// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

extern "C" __global__ void ComputeTotient(unsigned long long int N){

    int thr, size, rank;
    unsigned long long int count;

    thr = threadIdx.x;
    size = gridDim.x*NT;
    rank = blockIdx.x*NT + thr;

    count = 0;
    int temp;

    for (unsigned long long int i = rank; i < N; i += size){
        while (rank != 0) {
            temp = rank;
            rank = p % rank;
            p = temp;
        }
        if(p==1)
        ++count;
    }

 // Shared memory parallel reduction within thread block.
    shrCount[thr] = count;
    syncthreads();

    for (int i = NT/2; i > 0; i >>= 1){

        if (thr < i)
        shrCount[thr] += shrCount[thr+i];
        syncthreads();
    }

    // Atomic reduction into overall counter.
    if (thr == 0)
    atomicAdd (&devCount, shrCount[0]);

}
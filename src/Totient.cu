
#include <hip/hip_runtime.h>
// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

extern "C" __global__ void ComputeTotient(unsigned long long int P,
                                              unsigned long long int Q)
       {

    int thr, size, rank;
    thr = threadIdx.x;
    size = gridDim.x*NT;
    rank = blockIdx.x*NT + thr;



       }
#include "hip/hip_runtime.h"
// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

extern "C" __global__ void ComputeTotient(unsigned long long int N){

    int thr, size, rank;
    unsigned long long int count=0, b, temp, a;

    thr = threadIdx.x;
    size = gridDim.x*NT;
    rank = blockIdx.x*NT + thr;

    for (unsigned long long int x = rank; x < N; x += size){
    a=x;
    b=N;
        while (b > 0){
            temp = b;
            b = a % b;
            a = temp;
        }
        if(a==1)
        ++count;
    }

 // Shared memory parallel reduction within thread block.
    shrCount[thr] = count;
    syncthreads();

    for (int i = NT/2; i > 0; i >>= 1){

        if (thr < i)
        shrCount[thr] += shrCount[thr+i];
        syncthreads();
    }

    // Atomic reduction into overall counter.
    if (thr == 0)
    atomicAdd (&devCount, shrCount[0]);

}
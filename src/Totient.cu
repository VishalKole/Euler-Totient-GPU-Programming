#include "hip/hip_runtime.h"
// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

extern "C" __global__ void ComputeTotient(unsigned long long int P, unsigned long long int N){

    int thr, size, rank;
    unsigned long long int count;

    thr = threadIdx.x;
    size = gridDim.x*NT;
    rank = blockIdx.x*NT + thr;


 count = 0;

    int temp;
    for (unsigned long long int i = rank; i < N; i += size){
            while (q != 0) {
                temp = q;
                q = p % q;
                p = temp;
            }
            if(p==1)
            ++count;
}

 // Shared memory parallel reduction within thread block.
   shrCount[thr] = count;
   __syncthreads();

   for (int i = NT/2; i > 0; i >>= 1)
         {
         if (thr < i)
            shrCount[thr] += shrCount[thr+i];
         __syncthreads();
         }

            // Atomic reduction into overall counter.
            if (thr == 0)
               atomicAdd (&devCount, shrCount[0]);




       }